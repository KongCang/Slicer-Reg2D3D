#include "hip/hip_runtime.h"
/*
 *
 *  Insert comment here
 *
 */

#define DEBUG 0

// CUDA raycasting

#ifndef _VOLUMERENDERCUSTOM_KERNEL_CU_
#define _VOLUMERENDERCUSTOM_KERNEL_CU_

#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>

typedef unsigned int  uint;
typedef unsigned char uchar;

// output images
__constant__ unsigned short *image_raw_d_us;
__constant__ unsigned short *image_raw_d_mask1_us;
__constant__ unsigned short *image_raw_d_mask2_us;

// volume array and texture
hipArray *d_volumeArray = 0;
texture<unsigned short, 3, hipReadModeNormalizedFloat> texUS;

// mask arrays and textures for cam 1 & 2
hipArray *d_mask1Array = 0;
texture<unsigned char, 2, hipReadModeElementType> texMask1;

hipArray *d_mask2Array = 0;
texture<unsigned char, 2, hipReadModeElementType> texMask2;

// transfer function arrays for cam 1 & 2
hipArray *d_transferFuncArray = 0;
texture<float, 1, hipReadModeElementType> transferTex; // 1D transfer function texture

hipArray *d_transferFuncArray_cam2 = 0;
texture<float, 1, hipReadModeElementType> transferTexCam2; // 1D transfer function texture


unsigned short width, height, depth;
long voxel_number;


typedef struct {
    float4 m[3];
} float3x4;

__constant__ float3x4 myInvModelViewMatrix;

struct Ray {
        float3 o;	// origin
	float3 d;	// direction
};

// intersect ray with a box
// http://www.siggraph.org/education/materials/HyperGraph/raytrace/rtinter3.htm

__device__
int intersectBox(Ray r, float3 boxmin, float3 boxmax, float *tnear, float *tfar)
{
    // compute intersection of ray with all six bbox planes
    float3 invR = make_float3(1.0f) / r.d;
    float3 tbot = invR * (boxmin - r.o);
    float3 ttop = invR * (boxmax - r.o);

    // re-order intersections to find smallest and largest on each axis
    float3 tmin = fminf(ttop, tbot);
    float3 tmax = fmaxf(ttop, tbot);

    // find the largest tmin and the smallest tmax
    float largest_tmin = fmaxf(fmaxf(tmin.x, tmin.y), fmaxf(tmin.x, tmin.z));
    float smallest_tmax = fminf(fminf(tmax.x, tmax.y), fminf(tmax.x, tmax.z));

	*tnear = largest_tmin;
	*tfar = smallest_tmax;

	return smallest_tmax > largest_tmin;
}

// transform vector by matrix (no translation)
__device__
float3 mul(const float3x4 &M, const float3 &v)
{
    float3 r;
    r.x = dot(v, make_float3(M.m[0]));
    r.y = dot(v, make_float3(M.m[1]));
    r.z = dot(v, make_float3(M.m[2]));
    return r;

}

// transform vector by matrix with translation
__device__
float4 mul(const float3x4 &M, const float4 &v)
{
    float4 r;
    r.x = dot(v, M.m[0]);
    r.y = dot(v, M.m[1]);
    r.z = dot(v, M.m[2]);
    r.w = 1.0f;
    return r;
}

void checkCUDAError(const char* msg) {
hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    fprintf(stderr, "Cuda error(%d): %s: %s.\n", err, msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
#if DEBUG
  else
      printf("%s - OK.\n", msg);
#endif
}

__global__ void
d_renderUS(unsigned short *d_output, uint imageSize, float focal_width, float imagePixelSpacing, uint minX, uint maxX, uint minY, uint maxY, int intensityDivider, short currentMask)
{
    const int maxSteps = 50000;
    const float tstep = 0.00606f;
    const float3 boxMin = make_float3(-1.0f, -1.0f, -1.0f);
    const float3 boxMax = make_float3(1.0f, 1.0f, 1.0f);


    uint x = blockIdx.x*blockDim.x + threadIdx.x;
    uint y = blockIdx.y*blockDim.y + threadIdx.y;

    if (x >= imageSize || y >= imageSize)
        return;

    // scale the coordinates to be between -1 and 1
    float u = (x / (float) imageSize)*2.0f-1.0f;
    float v = (y / (float) imageSize)*2.0f-1.0f;

    // calculate eye ray in world space
    Ray eyeRay;
    eyeRay.o = make_float3(mul(myInvModelViewMatrix, make_float4(0.0f, 0.0f, 0.0f, 1.0f)));
    //eyeRay.d = normalize(make_float3(u, v, -1.0f * ( (410.0f / 264.0f) * 2.0*focal_width/(float)imageSize)/imagePixelSpacing)); // ((float)nVoxX/(voxSpacingZ*(float)nVoxZ))
    eyeRay.d = normalize(make_float3(u, v, -1.0f * ( 2.0*focal_width/(float)imageSize)/imagePixelSpacing)); // ((float)nVoxX/(voxSpacingZ*(float)nVoxZ))
    eyeRay.d = mul(myInvModelViewMatrix, eyeRay.d);
    //eyeRay.d.z = eyeRay.d.z * 410.0f / 264.0f; // FIXME: all these values are hardcoded, must now fix this
    eyeRay.d.z = eyeRay.d.z; // FIXME: all these values are hardcoded, must now fix this

    // find intersection with box
    float tnear, tfar;
    int hit = intersectBox(eyeRay, boxMin, boxMax, &tnear, &tfar);
    if (!hit) return;
    if (tnear < 0.0f) tnear = 0.0f;     // clamp to near plane

    // march along ray from front to back, accumulating color
    //float4 sum = make_float4(0.0f);
    float unisum = 0;
    float t = tnear;
    float3 pos = eyeRay.o + eyeRay.d*tnear;
    float3 step = eyeRay.d*tstep;
    //step.z = step.z * 410.0f / 264.0f;
    step.z = step.z;

    for(int i=0; i<maxSteps; i++) {

        float sample = tex3D(texUS, pos.x*0.5f+0.5f, pos.y*0.5f+0.5f, pos.z*0.5f+0.5f);

        unisum += sample/(float)intensityDivider;

        t += tstep;
        if (t > tfar) break;

        pos += step;
    }

    if(currentMask == 0)    // rendering cam1
        unisum = tex1D(transferTex, unisum) * 65535;
    else                    // rendering cam2
        unisum = tex1D(transferTexCam2, unisum) * 65535;

    // write to the output buffer
    d_output[y*imageSize + x] = ushort(unisum);

}

__global__ void
d_renderUS_mask1(unsigned short *d_output, uint imageSize, float focal_width, float imagePixelSpacing, uint minX, uint maxX, uint minY, uint maxY, int intensityDivider)
{
    const int maxSteps = 50000;
    const float tstep = 0.00606f;
    const float opacityThreshold = 242.0f; // 0.95 * 255
    const float3 boxMin = make_float3(-1.0f, -1.0f, -1.0f);
    const float3 boxMax = make_float3(1.0f, 1.0f, 1.0f);


    uint x = blockIdx.x*blockDim.x + threadIdx.x + minX;
    uint y = blockIdx.y*blockDim.y + threadIdx.y + minY;

    if( tex2D(texMask1, x, y) == uchar(0) )
        return;

    // scale the coordinates to be between -1 and 1
    float u = (x / (float) imageSize)*2.0f-1.0f;
    float v = (y / (float) imageSize)*2.0f-1.0f;

    // calculate eye ray in world space
    Ray eyeRay;
    eyeRay.o = make_float3(mul(myInvModelViewMatrix, make_float4(0.0f, 0.0f, 0.0f, 1.0f)));
    //eyeRay.o = make_float3(mul(myInvModelViewMatrix, make_float4(u, v, -1.0f * (focal_width/(float)imageSize), 1.0f))); // cast from the screen instead from 0
    eyeRay.d = normalize(make_float3(u, v, -1.0f * (2.0*focal_width/(float)imageSize)/imagePixelSpacing));
    eyeRay.d = mul(myInvModelViewMatrix, eyeRay.d);

    // find intersection with box
    float tnear, tfar;
    int hit = intersectBox(eyeRay, boxMin, boxMax, &tnear, &tfar);
    if (!hit) return;
    if (tnear < 0.0f) tnear = 0.0f;     // clamp to near plane

    // march along ray from front to back, accumulating color
    float4 sum = make_float4(0.0f);
    float unisum = 0;
    float t = tnear;
    float3 pos = eyeRay.o + eyeRay.d*tnear;
    float3 step = eyeRay.d*tstep;

    //float sample = 0;

    for(int i=0; i<maxSteps; i++) {

        float sample = tex3D(texUS, pos.x*0.5f+0.5f, pos.y*0.5f+0.5f, pos.z*0.5f+0.5f);

        unisum += sample/(float)intensityDivider;

        // exit early if opaque
        //if (unisum > (opacityThreshold))
        //    break;

        t += tstep;
        if (t > tfar) break;

        pos += step;
    }

    unisum = tex1D(transferTex, unisum) * 65535;

    // write to the output buffer
    d_output[y*imageSize + x] = ushort(unisum);

    // Test pattern.
    //d_output[y*imageSize + x] = uchar( ((x/(float)imageSize)/2 + (y/(float)imageSize)/2 )*255);
}

__global__ void
d_renderUS_mask2(unsigned short *d_output, uint imageSize, float focal_width, float imagePixelSpacing, uint minX, uint maxX, uint minY, uint maxY, int intensityDivider)
{
    const int maxSteps = 50000;
    const float tstep = 0.00606f;
    const float opacityThreshold = 242.0f; // 0.95 * 255
    const float3 boxMin = make_float3(-1.0f, -1.0f, -1.0f);
    const float3 boxMax = make_float3(1.0f, 1.0f, 1.0f);


    uint x = blockIdx.x*blockDim.x + threadIdx.x + minX;
    uint y = blockIdx.y*blockDim.y + threadIdx.y + minY;

    if( tex2D(texMask2, x, y) == uchar(0) )
        return;

    // scale the coordinates to be between -1 and 1
    float u = (x / (float) imageSize)*2.0f-1.0f;
    float v = (y / (float) imageSize)*2.0f-1.0f;

    // calculate eye ray in world space
    Ray eyeRay;
    eyeRay.o = make_float3(mul(myInvModelViewMatrix, make_float4(0.0f, 0.0f, 0.0f, 1.0f)));
    //eyeRay.o = make_float3(mul(myInvModelViewMatrix, make_float4(u, v, -1.0f * (focal_width/(float)imageSize), 1.0f))); // cast from the screen instead from 0
    eyeRay.d = normalize(make_float3(u, v, -1.0f * (2.0*focal_width/(float)imageSize)/imagePixelSpacing));
    eyeRay.d = mul(myInvModelViewMatrix, eyeRay.d);

    // find intersection with box
    float tnear, tfar;
    int hit = intersectBox(eyeRay, boxMin, boxMax, &tnear, &tfar);
    if (!hit) return;
    if (tnear < 0.0f) tnear = 0.0f;     // clamp to near plane

    // march along ray from front to back, accumulating color
    float4 sum = make_float4(0.0f);
    float unisum = 0;
    float t = tnear;
    float3 pos = eyeRay.o + eyeRay.d*tnear;
    float3 step = eyeRay.d*tstep;

    //float sample = 0;

    for(int i=0; i<maxSteps; i++) {

        float sample = tex3D(texUS, pos.x*0.5f+0.5f, pos.y*0.5f+0.5f, pos.z*0.5f+0.5f);

        unisum += sample/(float)intensityDivider;

        // exit early if opaque
        //if (unisum > (opacityThreshold))
        //    break;

        t += tstep;
        if (t > tfar) break;

        pos += step;
    }

    unisum = tex1D(transferTexCam2, unisum) * 65535;

    // write to the output buffer
    d_output[y*imageSize + x] = ushort(unisum);

    // Test pattern.
    //d_output[y*imageSize + x] = uchar( ((x/(float)imageSize)/2 + (y/(float)imageSize)/2 )*255);
}

__global__ void
d_renderSliceUS(unsigned short *d_output, uint imageSize, bool useMask1, uint minX, uint maxX, uint minY, uint maxY, int intensityDivider)
{
    const float3 boxMin = make_float3(-1.0f, -1.0f, -1.0f);
    const float3 boxMax = make_float3(1.0f, 1.0f, 1.0f);

    uint x = blockIdx.x*blockDim.x + threadIdx.x;
    uint y = blockIdx.y*blockDim.y + threadIdx.y;

    // scale the coordinates to be between -1 and 1
    float u = (x / (float) imageSize)*2.0f-1.0f;
    float v = (y / (float) imageSize)*2.0f-1.0f;

    // calculate eye ray in world space
    Ray eyeRay;
    eyeRay.o = make_float3(mul(myInvModelViewMatrix, make_float4(u, v, 0.0f, 1.0f)));
    eyeRay.d = normalize(make_float3(u, v, 0));
    eyeRay.d = mul(myInvModelViewMatrix, eyeRay.d);

    // find intersection with box
    float tnear, tfar;
    int hit = intersectBox(eyeRay, boxMin, boxMax, &tnear, &tfar);
    if (!hit) return;
    if (tnear < 0.0f) tnear = 0.0f;     // clamp to near plane

    // march along ray from front to back, accumulating color
    //float3 pos = eyeRay.o + eyeRay.d*tnear;
    float3 pos = eyeRay.o;

    float sample = tex3D(texUS, pos.x*0.5f+0.5f, pos.y*0.5f+0.5f, pos.z*0.5f+0.5f);

    float unisum = tex1D(transferTex, sample) * 65535;

    // write to the output buffer
    d_output[y*imageSize + x] = ushort(unisum);
}

__global__ void
d_renderSliceUS_mask1(unsigned short *d_output, uint imageSize, bool useMask1, uint minX, uint maxX, uint minY, uint maxY, int intensityDivider)
{
    const float3 boxMin = make_float3(-1.0f, -1.0f, -1.0f);
    const float3 boxMax = make_float3(1.0f, 1.0f, 1.0f);

    uint x = blockIdx.x*blockDim.x + threadIdx.x + minX;
    uint y = blockIdx.y*blockDim.y + threadIdx.y + minY;

    if( tex2D(texMask1, x, y) == uchar(0) )
        return;

    // scale the coordinates to be between -1 and 1
    float u = (x / (float) imageSize)*2.0f-1.0f;
    float v = (y / (float) imageSize)*2.0f-1.0f;

    // calculate eye ray in world space
    Ray eyeRay;
    eyeRay.o = make_float3(mul(myInvModelViewMatrix, make_float4(u, v, 0.0f, 1.0f)));
    eyeRay.d = normalize(make_float3(u, v, 0));
    eyeRay.d = mul(myInvModelViewMatrix, eyeRay.d);

    // find intersection with box
    float tnear, tfar;
    int hit = intersectBox(eyeRay, boxMin, boxMax, &tnear, &tfar);
    if (!hit) return;
    if (tnear < 0.0f) tnear = 0.0f;     // clamp to near plane

    // march along ray from front to back, accumulating color
    float3 pos = eyeRay.o + eyeRay.d*tnear;



    float unisum = tex3D(texUS, pos.x*0.5f+0.5f, pos.y*0.5f+0.5f, pos.z*0.5f+0.5f);

    unisum = tex1D(transferTex, unisum) * 65535;

    // write to the output buffer
    d_output[y*imageSize + x] = ushort(unisum);
}

void drrRendererRayCastingCuda_updateTransferFunction(float *transferFunc)
{
    size_t tFunc_size = 12;
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();

    hipMallocArray( &d_transferFuncArray, &channelDesc, tFunc_size, 1);

    hipMemcpyToArray( d_transferFuncArray, 0, 0, transferFunc, tFunc_size*sizeof(float), hipMemcpyHostToDevice);

    transferTex.filterMode = hipFilterModeLinear;
    transferTex.normalized = true;    // access with normalized texture coordinates
    transferTex.addressMode[0] = hipAddressModeClamp;   // wrap texture coordinates

    // Bind the array to the texture
    hipBindTextureToArray( transferTex, d_transferFuncArray, channelDesc);
}

void drrRendererRayCastingCuda_updateTransferFunctionCam2(float *transferFunc)
{
    size_t tFunc_size = 12;
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();

    hipMallocArray( &d_transferFuncArray_cam2, &channelDesc, tFunc_size, 1);

    hipMemcpyToArray( d_transferFuncArray_cam2, 0, 0, transferFunc, tFunc_size*sizeof(float), hipMemcpyHostToDevice);

    transferTexCam2.filterMode = hipFilterModeLinear;
    transferTexCam2.normalized = true;    // access with normalized texture coordinates
    transferTexCam2.addressMode[0] = hipAddressModeClamp;   // wrap texture coordinates

    // Bind the array to the texture
    hipBindTextureToArray( transferTexCam2, d_transferFuncArray_cam2, channelDesc);
}

//===================================================================
//
// New copy function to CUDA arrays
//
//===================================================================
void drrRendererRayCastingCuda_prepareVolumeUS(unsigned short imageSize,
    unsigned short* volume, int volumeWidth, int volumeHeight,
    int volumeDepth, float* additionalParameters, unsigned char additionalParameterLength)
{
    #if DEBUG
        printf("drrRendererRayCastingCuda_prepareVolumeUS() called...\n");
    #endif

    checkCUDAError("Before Preparing volume");

    // initialize data and pointer
    voxel_number = (volumeWidth * volumeHeight * volumeDepth);
    size_t image_size = (imageSize * imageSize) * sizeof(unsigned short);

    hipExtent extentvolumeSize = make_hipExtent(volumeWidth, volumeHeight, volumeDepth);
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<unsigned short>();

    hipMalloc3DArray(&d_volumeArray, &channelDesc, extentvolumeSize);

    checkCUDAError("After malloc");

    // copy data to 3D array
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr   = make_hipPitchedPtr(volume, extentvolumeSize.width*sizeof(unsigned short), extentvolumeSize.width, extentvolumeSize.height);
    copyParams.dstArray = d_volumeArray;
    copyParams.extent   = extentvolumeSize;
    copyParams.kind     = hipMemcpyHostToDevice;
    hipMemcpy3D(&copyParams);

    // set texture parameters
    texUS.normalized = true;                      // access with normalized texture coordinates
    texUS.filterMode = hipFilterModeLinear;      // linear interpolation
    texUS.addressMode[0] = hipAddressModeClamp;  // clamp texture coordinates
    texUS.addressMode[1] = hipAddressModeClamp;

    // bind array to 3D texture
    hipBindTextureToArray(texUS, d_volumeArray, channelDesc);

    checkCUDAError("After binding texture");

    hipMalloc((void **) &image_raw_d_us, image_size);
    hipMemset(image_raw_d_us, 0, image_size);

    hipMalloc((void **) &image_raw_d_mask1_us, image_size);
    hipMemset(image_raw_d_mask1_us, 0, image_size);

    hipMalloc((void **) &image_raw_d_mask2_us, image_size);
    hipMemset(image_raw_d_mask2_us, 0, image_size);

    checkCUDAError("After preparing volume");

}

void drrRendererRayCastingCuda_prepareMask1(unsigned int xSize, unsigned int ySize, unsigned char *datatex)
{

#if DEBUG
    printf("drrRendererRayCastingCuda_prepareMask1() called...\n");
#endif

    hipMemset(image_raw_d_mask1_us, 0, xSize*ySize*sizeof(unsigned short));

    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<unsigned char>();
    hipMallocArray(&d_mask1Array, &channelDesc, xSize, ySize);

    hipMemcpyToArray(d_mask1Array, 0, 0, datatex, xSize*ySize, hipMemcpyHostToDevice);

    // set texture parameters
    texMask1.normalized = false;                      // access with normalized texture coordinates
    texMask1.filterMode = hipFilterModePoint;      // linear interpolation
    texMask1.addressMode[0] = hipAddressModeWrap;  // clamp texture coordinates
    texMask1.addressMode[1] = hipAddressModeWrap;
    hipBindTextureToArray(texMask1, d_mask1Array, channelDesc);

    #if DEBUG
        hipError_t err = hipGetLastError();
        if (err != 0)
        {
            printf("Error creating texture, code is %d\n", err);
            exit(1);
        }
    #endif
}

void drrRendererRayCastingCuda_prepareMask2(unsigned int xSize, unsigned int ySize, unsigned char *datatex)
{

#if DEBUG
    printf("drrRendererRayCastingCuda_prepareMask2() called...\n");
#endif

    hipMemset(image_raw_d_mask2_us, 0, xSize*ySize*sizeof(unsigned short));

    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<unsigned char>();
    hipMallocArray(&d_mask2Array, &channelDesc, xSize, ySize);

    hipMemcpyToArray(d_mask2Array, 0, 0, datatex, xSize*ySize, hipMemcpyHostToDevice);

    // set texture parameters
    texMask1.normalized = false;                      // access with normalized texture coordinates
    texMask1.filterMode = hipFilterModePoint;      // linear interpolation
    texMask1.addressMode[0] = hipAddressModeWrap;  // clamp texture coordinates
    texMask1.addressMode[1] = hipAddressModeWrap;
    hipBindTextureToArray(texMask2, d_mask2Array, channelDesc);

    #if DEBUG
        hipError_t err = hipGetLastError();
        if (err != 0)
        {
            printf("Error creating texture, code is %d\n", err);
            exit(1);
        }
    #endif
}

void drrRendererRayCastingCuda_computeMatrix(float *invModelViewMatrix, float focalWith)
{
    #if DEBUG
        printf("drrRendererRayCastingCuda_computeMatrix() called...\n");
    #endif

#if DEBUG
    checkCUDAError("Before copying matrix");
#endif

    // copy transformation data to board
    size_t inv_transformation_data_size = (12) * sizeof(float);
    hipMemcpyToSymbol(HIP_SYMBOL(myInvModelViewMatrix), invModelViewMatrix, inv_transformation_data_size);

#if DEBUG
    checkCUDAError("After copying matrix");
#endif

}

void drrRendererRayCastingCuda_free()
{
    #if DEBUG
        printf("drrRendereRayCastingCuda_free() called...\n");
    #endif

    // free all cuda pointers
    hipFreeArray(d_volumeArray);
    hipFreeArray(d_transferFuncArray);
    hipFree(image_raw_d_us);
    hipFree(image_raw_d_mask1_us);
    hipFree(image_raw_d_mask2_us);

    hipFreeArray(d_mask1Array);
    hipFreeArray(d_mask2Array);

    checkCUDAError("After freeing memory");

}

void render_kernelUS(dim3 gridSize, dim3 blockSize, unsigned short *d_output, uint imageSize, float imagePixelSpacing, float focal_width, bool useMask1, bool useMask2, uint minX, uint maxX, uint minY, uint maxY, int intensityDivider, short currentMask)
{
    #if DEBUG
        printf("render_kernelUS() called...\n");
        checkCUDAError("Before render kernel");
    #endif

    if(useMask1 && currentMask == 0) // Using mask1 and rendering Cam1
    {
        d_renderUS_mask1<<<gridSize, blockSize>>>( image_raw_d_mask1_us, imageSize, focal_width, imagePixelSpacing, minX, maxX, minY, maxY, intensityDivider);
        hipMemcpy(d_output+(minY)*imageSize, image_raw_d_mask1_us+(minY)*imageSize, sizeof(unsigned short)*imageSize*(maxY-minY+1), hipMemcpyDeviceToHost);
    }
    else if(useMask2 && currentMask == 1) // Using mask2 and rendering Cam2
    {
        d_renderUS_mask2<<<gridSize, blockSize>>>( image_raw_d_mask2_us, imageSize, focal_width, imagePixelSpacing, minX, maxX, minY, maxY, intensityDivider);
        hipMemcpy(d_output+(minY)*imageSize, image_raw_d_mask2_us+(minY)*imageSize, sizeof(unsigned short)*imageSize*(maxY-minY+1), hipMemcpyDeviceToHost);
    }
    else // Rendering Cam1 or Cam2 without any mask
    {
        d_renderUS<<<gridSize, blockSize>>>( image_raw_d_us, imageSize, focal_width, imagePixelSpacing, minX, maxX, minY, maxY, intensityDivider, currentMask);
        hipMemcpy(d_output, image_raw_d_us, sizeof(unsigned short)*imageSize*imageSize, hipMemcpyDeviceToHost);
    }

    #if DEBUG
        checkCUDAError("After render kernel");
    #endif

}

void renderSlice_kernelUS(dim3 gridSize, dim3 blockSize, unsigned short *d_output, uint imageSize, bool useMask1, uint minX, uint maxX, uint minY, uint maxY, int intensityDivider)
{
    #if DEBUG
        printf("renderSlice_kernelUS() called...\n");
        checkCUDAError("Before render kernel");
    #endif

    if(useMask1) // Rendering with a mask
    {
        d_renderSliceUS_mask1<<<gridSize, blockSize>>>( image_raw_d_mask1_us, imageSize, useMask1, minX, maxX, minY, maxY, intensityDivider);
        hipMemcpy(d_output+(minY)*imageSize, image_raw_d_mask1_us+(minY)*imageSize, sizeof(unsigned short)*imageSize*(maxY-minY+1), hipMemcpyDeviceToHost);
    }
    else // Rendering without a mask
    {
        d_renderSliceUS<<<gridSize, blockSize>>>( image_raw_d_us, imageSize, useMask1, minX, maxX, minY, maxY, intensityDivider);
        hipMemcpy(d_output, image_raw_d_us, sizeof(unsigned short)*imageSize*imageSize, hipMemcpyDeviceToHost);
    }

    #if DEBUG
        checkCUDAError("After slice render kernel");
    #endif

}

#endif // #ifndef _VOLUMERENDER_KERNEL_CU_
