// This is the REAL "hello world" for CUDA!
// It takes the string "Hello ", prints it, then passes it to CUDA with an array
// of offsets. Then the offsets are added in parallel to produce the string "World!"
// By Ingemar Ragnemalm 2010
 

#include <hip/hip_runtime.h>
#include <stdio.h>
 
const int N = 16; 
const int blocksize = 16; 
 
__global__ 
void hello(char *a, int *b) 
{
	a[threadIdx.x] += b[threadIdx.x];
}
 

void cuda_hello(char *ac, int* bc, int n)
{
//	char a[N] = "Hello \0\0\0\0\0\0";
//	int b[N] = {15, 10, 6, 0, -11, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
 
	char *ad;
	int *bd;
	const int csize = n*sizeof(char);
	const int isize = n*sizeof(int);
 
	printf("C:%s --> ", ac);
 
	hipMalloc( (void**)&ad, csize ); 
	hipMalloc( (void**)&bd, isize ); 
	hipMemcpy( ad, ac, csize, hipMemcpyHostToDevice ); 
	hipMemcpy( bd, bc, isize, hipMemcpyHostToDevice ); 
	
	dim3 dimBlock( n, 1 );
	dim3 dimGrid( 1, 1 );
        hello<<<dimGrid, dimBlock>>>(ad, bd);
//	hello<<<N, 1>>>(ad, bd);
	hipMemcpy( ac, ad, csize, hipMemcpyDeviceToHost ); 
	hipFree( ad );
	hipFree( bd );
	
	printf("C: %s\n", ac);

}

